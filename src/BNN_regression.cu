#include "hip/hip_runtime.h"
/*!****************************************
  \file BNN_regression.cc
 
  \author Michelle E. Perry
  \date created 25/2/2013 updated 9/9/2013
*******************************************/

#include "BNN_regression.cuh"
#include <math.h>
#include <iostream>
#include <iomanip>
#include <cassert>
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h> /// thrust parallel reduce
#include <thrust/device_free.h>

/*!**
NN PARAMETERS
q[0] = b;
q[j+1] = v_j;
q[H+j+1] = a_j
q[2H+1+i+j*I] = u_ji
 ****/


__global__ void diff(HMC_type* q, HMC_type* x, HMC_type* w, HMC_type* t, HMC_type* d, int H, int I, int N) {

  int th = threadIdx.x + blockIdx.x*blockDim.x;  
  if(th < N){
  HMC_type f = q[0];
  for(int j=0;j<H;j++){
    HMC_type inSum = 0.0;
    for(int i=0;i<I;i++){
      inSum += q[2*H+1+I*j+i]*x[th*I+i];
    }
    f+=q[j+1]*tanh(q[H+1+j]+inSum);
  }
  d[th] = w[th]*(t[th]-f)*(t[th]-f);
  }
};
__global__ void diff2(HMC_type* q, HMC_type* x, HMC_type* w, HMC_type* t, HMC_type* d, int H, int I, HMC_type* block_sum, int N) {


  int tid = threadIdx.x;
  int th = threadIdx.x + blockIdx.x*blockDim.x;  
  if(th < N ){
  extern __shared__ HMC_type sdata[];
  HMC_type f = q[0];
  for(int j=0;j<H;j++){
    HMC_type inSum = 0.0;
    for(int i=0;i<I;i++){
      inSum += q[2*H+1+I*j+i]*x[th*I+i];
    }
    f+=q[j+1]*tanh(q[H+1+j]+inSum);
  }
  d[th] = w[th]*(t[th]-f)*(t[th]-f);
  /* parallel reduction */
  sdata[tid] = d[th];
  __syncthreads();

  //do reduction in shared memory.
  for(unsigned int s=1;s<blockDim.x;s*=2){
    //if(tid < s){
    //sdata[tid] += sdata[tid+s];
    //}
    int index = 2*s*tid;
    if(index < blockDim.x){
      sdata[index] += sdata[index+s];
    }
    //if(tid % (2*s) == 0){
    //sdata[tid] += sdata[tid+s];
    //}
    __syncthreads();
  }
  if(tid ==0) block_sum[blockIdx.x]=sdata[0];
  }
};
 HMC_type diff3(std::vector<HMC_type> &q, std::vector<HMC_type> &x, std::vector<HMC_type> &w, std::vector<HMC_type> &t, int H, int I, int N) {

  HMC_type d = 0.0;
  for(int th=0;th<N;th++){
    HMC_type f = q[0];
    //  std::cout << "here" << std::endl;
    for(int j=0;j<H;j++){
      HMC_type inSum = 0.0;
      for(int i=0;i<I;i++){
	inSum += q[2*H+1+I*j+i]*x[th*I+i];//****CHANGED****//
      }
      f+=q[j+1]*tanh(q[H+1+j]+inSum);//********//
    }
    d += w[th]*(t[th]-f)*(t[th]-f);
    if(th%256 == 0){
      std::cout << "!!!!!!!!" << std::endl;
      std::cout << d << std::endl;
    }
  }
  return d;
}

inline void cudaCheckError(const char * fn) 
{
    const hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        std::cerr << fn << ": " << hipGetErrorString(err) << "!\n";
        std::exit(EXIT_FAILURE);
    }
}


BNN_regression::BNN_regression(int l, int nOut_, int nRep_, int nBurn_, int h, int inp, std::vector<HMC_type> &data, std::vector<HMC_type> &weights, std::vector<HMC_type> &targets, std::string s) 
  : HMC_base(l, 1+h*(2+inp), nOut_, nRep_, nBurn_),
    v_x(data),
    H(h),
    I(inp),
    N(targets.size()),
    sig(1),
    v_w(weights),
    v_t(targets),
    oFile(s),
    sigb(100),
    sigv(1.65),
    siga(0.86),
    sigu(0.54)
{
  std::cout << "N = " << N << std::endl;
  np = getNP();
  std::cout << "BNN_regression(L,nOut,nRep,nBurn,H,I,x,w,t,s)" << std::endl;
  std::cout << "v_x[0] = " << v_x[0] << "\t v_x[1] = " << v_x[1] << std::endl;
  std::cout << "v_t[0] = " << v_t[0] << std::endl;
  of.open(oFile.c_str(),std::ofstream::out);
  of << I << "\t" << H << std::endl;

  /** memory allocation on GPU **/
  /* need to allocate d,t,x,w,q  */
  size_np = np*sizeof(HMC_type);
  size_n = N*sizeof(HMC_type);  

  //hipSetDevice(0);
  //hipDeviceSynchronize();
  //hipDeviceSynchronize();
  size_t freeMem, totalMem;

  hipMemGetInfo(&freeMem, &totalMem);

  fprintf(stderr, "Free = %ld, Total = %ld\n", freeMem, totalMem); 

  x = &v_x[0];
  t = &v_t[0];
  w = &v_w[0];
  //q = &v_q[0];
  //d = &v_d[0];
  
  /** allocate data x **/
  dev_x = new HMC_type[N*I];
  hipMalloc(&dev_x,size_n*I);
  cudaCheckError("hipMalloc dev_x");
  /** allocate dev targets t **/
  dev_t = new HMC_type[N];
  hipMalloc(&dev_t,size_n);
  cudaCheckError("hipMalloc dev_t");
  /** allocate dev weights w **/  
  dev_w = new HMC_type[N];
  hipMalloc(&dev_w,size_n);
  cudaCheckError("hipMalloc dev_w");
  /** allocate NN parameters q **/
  dev_q = new HMC_type[np];  
  hipMalloc(&dev_q,size_np);
  cudaCheckError("hipMalloc dev_q");
  /** allocate diff array **/  
  dev_d = new HMC_type[N];
  hipMalloc(&dev_d,size_n);
  cudaCheckError("hipMalloc dev_t");
  //  thrust::device_ptr<HMC_type> d_dev_ptr(dev_d);

  std::cout << "GPU memory allocated\n";

  d_dev_ptr = thrust::device_ptr<HMC_type>(dev_d);
  /** copying memory from CPU to GPU **/
  hipMemcpy(dev_x, x, size_n*I, hipMemcpyHostToDevice);
  cudaCheckError("hipMemcpy dev_x");
  hipMemcpy(dev_t, t, size_n, hipMemcpyHostToDevice);
  cudaCheckError("hipMemcpy dev_t");
  hipMemcpy(dev_w, w, size_n, hipMemcpyHostToDevice);
  cudaCheckError("hipMemcpy dev_w");
  std::cout << "GPU memory filled\n";
  
};

BNN_regression::BNN_regression(int l, int nOut_, int nRep_, int nBurn_, int h, int i, std::string s) 
  : HMC_base(l, 1+h*(2+i), nOut_, nRep_, nBurn_),
    v_x(std::vector<HMC_type>()),
    H(h),
    I(i),
    N(0),
    sig(1),
    v_w(std::vector<HMC_type>()),
    v_t(std::vector<HMC_type>()),
    oFile(s),
    sigb(100),
    sigv(1.65),
    siga(0.86),
    sigu(0.54)
{
  np = getNP();
  std::cout << "BNN_regression(l,h,i,s)" << std::endl;
  std::cout << "x[0] = " << x[0] << "\t x[1] = " << x[1] << std::endl;
  std::cout << "t[0] = " << t[0] << std::endl;
  of.open(oFile.c_str(),std::ofstream::out);

  of << I << "\t" << H << std::endl;

 /** memory allocation on GPU **/
  /* need to allocate d,t,x,w,q  */
  size_t size_np = np*sizeof(HMC_type);
  size_t size_n = N*sizeof(HMC_type);  

  /** allocate data x **/
  dev_x = new HMC_type[N*I];
  hipMalloc(&dev_x,size_n*I);
  /** allocate dev targets t **/
  dev_t = new HMC_type[N];
  hipMalloc(&dev_t,size_n);
  /** allocate dev weights w **/  
  dev_w = new HMC_type[N];
  hipMalloc(&dev_w,size_n);
  /** allocate NN parameters q **/
  dev_q = new HMC_type[np];  
  hipMalloc(&dev_q,size_np);
  /** allocate diff array **/  
  dev_d = new HMC_type[N];
  hipMalloc(&dev_d,size_n);

  /** copying memory from CPU to GPU **/
  hipMemcpy(dev_x, x, size_n*I, hipMemcpyHostToDevice);
  cudaCheckError("hipMemcpy dev_x");
  hipMemcpy(dev_t, t, size_n, hipMemcpyHostToDevice);
  cudaCheckError("hipMemcpy dev_t");
  hipMemcpy(dev_w, w, size_n, hipMemcpyHostToDevice);
  cudaCheckError("hipMemcpy dev_w");
};
void BNN_regression::add(std::vector<HMC_type> &inputs, HMC_type target, HMC_type weight){
  std::cout << "TO BE IMPLEMENTED" << std::endl;
};

BNN_regression::~BNN_regression() {
  std::cout << "destructor of BNN_regression()" << std::endl;
  hipFree(dev_x);
  hipFree(dev_t);
  hipFree(dev_w);
  hipFree(dev_q);
  hipFree(dev_d);

  thrust::device_free(d_dev_ptr);
};

inline HMC_type BNN_regression::U(std::vector<HMC_type> &in_q) {
  // if((int)q.size() != np){
  //std::cout << "ERROR: incorrect parameter numbers" << std::endl;
  //assert(0);
  // }
  /*! probability density for BNN regression is -logP */
  // HMC_type sum = 0;
  // for(int j=0;j<np;j++){
  //std::cout << q[j] << "\t";
  //}
  //std::cout << std::endl;
  //std::vector<HMC_type> diff(N);
  //thrust::host_vector<HMC_type> diff(N);
  HMC_type* qq = &in_q[0];
  int threadsPerBlock = 256;
  int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  
  hipMemcpy(dev_q, qq, size_np, hipMemcpyHostToDevice);
  cudaCheckError("hipMemcpy dev_q");
  //std::cout << "IN BNNREG" << std::endl;
  //thrust::device_ptr<HMC_type> d_dev_ptr(dev_d);
 
  diff<<<blocksPerGrid,threadsPerBlock>>>(dev_q, dev_x, dev_w, dev_t, dev_d, H, I, N); 
  //diff<<<threadsPerBlock,blocksPerGrid>>>(dev_q, dev_x, dev_w, dev_t, dev_d, H, I, N); 
  //HMC_type sum3 = diff3(in_q, v_x, v_w, v_t, H, I, N); 
  //diff[i] = w[i]*(t[i]-fq)*(t[i]-fq);
  //}
  HMC_type sum = thrust::reduce(d_dev_ptr,d_dev_ptr+N);
  //hipDeviceSynchronize();
  //HMC_type* dev_sum;// = new HMC_type[blocksPerGrid];
  //hipMalloc(&dev_sum,blocksPerGrid*sizeof(HMC_type));
  //cudaCheckError("hipMalloc dev_sum");
  //HMC_type* par_sum = new HMC_type[blocksPerGrid];
  //diff2<<<blocksPerGrid,threadsPerBlock>>>(dev_q, dev_x, dev_w, dev_t, dev_d, H, I, dev_sum, N); 
  //diff2<<<threadsPerBlock,blocksPerGrid>>>(dev_q, dev_x, dev_w, dev_t, dev_d, H, I, dev_sum, N); 
  //hipDeviceSynchronize();
  //hipMemcpy(par_sum, dev_sum, sizeof(HMC_type)*blocksPerGrid, hipMemcpyDeviceToHost);
  //cudaCheckError("hipMemcpy sum");
  
  //std::cout << blocksPerGrid << std::endl;
  //HMC_type sum2 = 0.0;
  //for(int i=0;i<blocksPerGrid;i++){
  //std::cout << par_sum[i] << "\t";
  //sum2 += par_sum[i];
  //}
  //std::cout << "IN BNNREG" << std::endl;
  // thrust::device_vector<HMC_type> diff_d = diff;
  //HMC_type sum = thrust::reduce(diff_d.begin(), diff_d.end());
  //std::cout << "GPU thrust sum = " << sum << "\t" << "GPU sum = " << sum2 << std::endl;//"\tCPU sum = " << sum3 << std::endl;
  //char test;
  //std::cin >> test; 

  return sum/(2*sig*sig) + LnPrior(in_q);

};
/*! OPTIMIZE! CACHE!!!!! */
inline std::vector<HMC_type> BNN_regression::delU(std::vector<HMC_type> &q) {
  std::vector<HMC_type> FDdU(np);// = new HMC_type[np];
  std::vector<HMC_type> qm(np);
  std::vector<HMC_type> qp(np);
  
  
  //std::cout<< "in BNN_regression::delU" << std::endl;
   HMC_type h = 0.001;
   //std::vector<HMC_type> dU(np);
   for(int i=0; i<np;i++){
     qm = q;
     qp = q;
     qm[i] = qm[i] - 0.5*h;
     qp[i] = qp[i] + 0.5*h;
     //std::cout<< qm[i] << "\t";
     //std::cout<< qp[i] << "\t";
     FDdU[i] =  (U(qp) - U(qm))/h;
   }
   //std::cout<<std::endl;
  
   // FDdU = FDdelU(q);
  return FDdU;  //int np = getNP();

};



inline HMC_type BNN_regression::LnPrior(std::vector<HMC_type> &q) {


  HMC_type prior = 0.0;
  prior += 0.5*q[0]*q[0]/(sigb*sigb);
  for (int v=1; v<H+1;v++){
    // std::cout << "v = " << v << std::endl;
    prior +=  0.5*q[v]*q[v]/(sigv*sigv);
  }
  for(int a=H+1;a<2*H+1;a++){
    prior +=  0.5*q[a]*q[a]/(siga*siga);
    //std::cout << "a = " << a << std::endl;
  }
  for(int u=2*H+1; u<H*(2+I)+1; u++){
    prior +=  0.5*q[u]*q[u]/(sigu*sigu);
    //std::cout << "u = " << u << std::endl;
  }
  return prior;
};

